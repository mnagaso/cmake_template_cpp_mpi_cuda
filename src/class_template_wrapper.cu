#include "hip/hip_runtime.h"
#include "class_template_wrapper.cuh"


// allocate memory on device
template <typename T>
void allocate_memory_on_device(T*& d_ptr, size_t size)
{
    hipMalloc(&d_ptr, size * sizeof(T));
}

// deallocate memory on device
template <typename T>
void deallocate_memory_on_device(T*& d_ptr)
{
    hipFree(d_ptr);
}

// copy memory from host to device
template <typename T>
void copy_host_to_device(T* d_ptr, T* h_ptr, size_t size)
{
    hipMemcpy(d_ptr, h_ptr, size * sizeof(T), hipMemcpyHostToDevice);
}

// copy memory from device to host
template <typename T>
void copy_device_to_host(T* h_ptr, T* d_ptr, size_t size)
{
    hipMemcpy(h_ptr, d_ptr, size * sizeof(T), hipMemcpyDeviceToHost);
}


// kernel function to add 2 to all elements of a vector
__global__ void add_2_to_all_elements_i_kernel(int* d_ptr, size_t size)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size)
        d_ptr[i] += 4;
}

// integer
void add_2_to_all_elements_i(int* d_ptr, size_t size)
{
    add_2_to_all_elements_i_kernel<<<size / BLOCK_SIZE, BLOCK_SIZE>>>(d_ptr, size);
}