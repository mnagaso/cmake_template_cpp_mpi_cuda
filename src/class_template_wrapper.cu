#include "hip/hip_runtime.h"
#include "class_template_wrapper.cuh"


// allocate memory on device
void allocate_memory_on_device_i(int*& d_ptr, size_t size)
{
    hipMalloc(&d_ptr, size * sizeof(int));
}

void allocate_memory_on_device_d(double*& d_ptr, size_t size)
{
    hipMalloc(&d_ptr, size * sizeof(double));
}

// deallocate memory on device
void deallocate_memory_on_device_i(int*& d_ptr)
{
    hipFree(d_ptr);
}
void deallocate_memory_on_device_d(double*& d_ptr)
{
    hipFree(d_ptr);
}



// copy memory from host to device
void copy_host_to_device_i(int* d_ptr, int* h_ptr, size_t size)
{
    hipMemcpy(d_ptr, h_ptr, size * sizeof(int), hipMemcpyHostToDevice);
}
void copy_host_to_device_d(double* d_ptr, double* h_ptr, size_t size)
{
    hipMemcpy(d_ptr, h_ptr, size * sizeof(double), hipMemcpyHostToDevice);
}




// copy memory from device to host
void copy_device_to_host_i(int* h_ptr, int* d_ptr, size_t size)
{
    hipMemcpy(h_ptr, d_ptr, size * sizeof(int), hipMemcpyDeviceToHost);
}
void copy_device_to_host_d(double* h_ptr, double* d_ptr, size_t size)
{
    hipMemcpy(h_ptr, d_ptr, size * sizeof(double), hipMemcpyDeviceToHost);
}


// kernel function to add 2 to all elements of a vector

// integer
__global__ void add_2_to_all_elements_i_kernel(int* d_ptr, size_t size)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size)
        d_ptr[i] += 2;
}

void add_2_to_all_elements_i(int* d_ptr, size_t size)
{
    add_2_to_all_elements_i_kernel<<<size / BLOCK_SIZE, BLOCK_SIZE>>>(d_ptr, size);
}

// double
__global__ void add_2_to_all_elements_d_kernel(double* d_ptr, size_t size)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size)
        d_ptr[i] += 2.0;
}

void add_2_to_all_elements_d(double* d_ptr, size_t size)
{
    add_2_to_all_elements_d_kernel<<<size / BLOCK_SIZE, BLOCK_SIZE>>>(d_ptr, size);
}